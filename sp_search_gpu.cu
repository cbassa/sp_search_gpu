#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hipfft/hipfft.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <getopt.h>

#define LIM 128

static __device__ __host__ inline hipfftComplex ComplexScale(hipfftComplex a,float s)
{
  hipfftComplex c;
  c.x=s*a.x;
  c.y=s*a.y;
  return c;
}

static __device__ __host__ inline hipfftComplex ComplexMul(hipfftComplex a,hipfftComplex b)
{
  hipfftComplex c;
  c.x=a.x*b.x-a.y*b.y;
  c.y=a.x*b.y+a.y*b.x;
  return c;
}

static __global__ void PointwiseComplexMultiply(hipfftComplex *a,hipfftComplex *b,hipfftComplex *c,int n,int m,int j,float scale)
{
  int i;
  const int numThreads=blockDim.x*gridDim.x;
  const int threadID=blockIdx.x*blockDim.x+threadIdx.x;

  for (i=threadID;i<n;i+=numThreads)
    c[i]=ComplexScale(ComplexMul(a[i],b[i%m+j*m]),scale);
}

__global__ void padd_data(float *y,float *ytmp,int nsamp,int nbin,int m,int nfft)
{
  int isamp,idx,ibin,ifft,ioverlap;

  // Indices of input data                                                                    
  ibin=blockIdx.x*blockDim.x+threadIdx.x;
  ifft=blockIdx.y*blockDim.y+threadIdx.y;

  // Compute valid threads
  if (ibin<nbin && ifft<nfft) {
    ioverlap=(nbin-m)/2;
    
    for (ifft=0;ifft<nfft;ifft++) {
      idx=ibin+nbin*ifft;
      if (idx<nbin*nfft) {
	isamp=ibin+m*ifft-ioverlap;
	if (isamp<0 || isamp>=nsamp)
	  ytmp[idx]=0.0;
	else
	  ytmp[idx]=y[isamp];
      }
    }
  }
  
  return;
}

__global__ void unpadd_data(float *y,float *ytmp,int n,int nx,int m,int ny)
{
  int i,j,k,l;
  int ioverlap;
  const int numThreads=blockDim.x*gridDim.x;
  const int threadID=blockIdx.x*blockDim.x+threadIdx.x;

  for (i=threadID;i<nx;i+=numThreads) {
    ioverlap=(nx-m)/2;
    
    for (j=0;j<ny;j++) {
      k=i+nx*j;
      if (k<nx*ny) {
	l=i+m*j-ioverlap;
	if (l>=0 && l<n && i>=ioverlap && i<m+ioverlap)
	  y[l]=ytmp[k];
      }
    }
  }

  return;
}

__global__ void prune(float *z,int n,int dw,int *mask,float sigma)
{
  int i,j;
  const int numThreads=blockDim.x*gridDim.x;
  const int threadID=blockIdx.x*blockDim.x+threadIdx.x;

  for (i=threadID;i<n;i+=numThreads) {
    mask[i]=1;
    if (z[i]<sigma) 
      mask[i]=0;
    for (j=i-dw/2;j<=i+dw/2;j++) {
      if (j<0 || j>=n)
	continue;
      if (z[j]<sigma)
	continue;
      if (z[j]>z[i])
	mask[i]=0;
    }
  }
  return;
}

__global__ void prune_final(float *z,int *dw,int *mask,int n,float sigma)
{
  int i,j;
  const int numThreads=blockDim.x*gridDim.x;
  const int threadID=blockIdx.x*blockDim.x+threadIdx.x;

  for (i=threadID;i<n;i+=numThreads) {
    mask[i]=1;

    // Mask candidates within half-width with lower significance
    if (z[i]>sigma) {
      for (j=i-dw[i]/2;j<=i+dw[i]/2;j++) {
	if (j<0 || j>=n)
	  continue;
	if (z[j]<z[i])
	  mask[j]=0;
      }
    }
  }

  return;
}

__global__ void store(float *x,float *z,int *mask,int *w,int wnew,int n)
{
  int i;
  const int numThreads=blockDim.x*gridDim.x;
  const int threadID=blockIdx.x*blockDim.x+threadIdx.x;

  for (i=threadID;i<n;i+=numThreads) {
    if (z[i]*mask[i]>x[i]) {
      x[i]=z[i]*mask[i];
      w[i]=wnew;
    }
  }

  return;
}

__global__ void store_final(float *x,int *mask,int n,float sigma)
{
  int i;
  const int numThreads=blockDim.x*gridDim.x;
  const int threadID=blockIdx.x*blockDim.x+threadIdx.x;

  for (i=threadID;i<n;i+=numThreads) {
    x[i]*=mask[i];
    if (x[i]<sigma) 
      x[i]=0.0;
  }

  return;
}

__global__ void detrend_and_normalize(float *y,float *ytmp,int nsamp,int mdetrend,int ndetrend,int ndm)
{
  int j,l,isamp,isamp0,isamp1;
  int idetrend,idm,idx;
  float x,s,sx,sxx,sy,sxy,syy,d,a,b;
  int isampmin,isampmax,lmax;
  float ymin,ymax,yswap,ystd;

  idetrend=blockIdx.x*blockDim.x+threadIdx.x;
  idm=blockIdx.y*blockDim.y+threadIdx.y;
  if (idetrend>=ndetrend || idm>=ndm)
    return;

  // Compute sums
  s=sx=sxx=sy=sxy=0.0;
  for (j=0;j<mdetrend;j++) {
    isamp=idetrend*mdetrend+j;
    idx=isamp+idm*nsamp;
    if (isamp>=nsamp)
      break;
    x=-0.5+(float) j/(float) mdetrend;
    s+=1.0;
    sx+=x;
    sxx+=x*x;
    sy+=y[idx];
    sxy+=x*y[idx];
  }

  // Linear parameters
  d=s*sxx-sx*sx;
  a=(sxx*sy-sx*sxy)/d;
  b=(s*sxy-sx*sy)/d;

  // Remove trend
  s=syy=0.0;
  for (j=0;j<mdetrend;j++) {
    isamp=idetrend*mdetrend+j;
    idx=isamp+idm*nsamp;
    if (isamp>=nsamp)
      break;
    x=-0.5+(float) j/(float) mdetrend;
    y[idx]-=a+b*x;
    ytmp[idx]=y[idx];
    s+=1.0;
    syy+=y[idx]*y[idx];
  }

  // Remove outliers 2.5% on either end
  isamp0=idetrend*mdetrend;
  isamp1=(idetrend+1)*mdetrend;
  lmax=mdetrend;
  if (isamp1>=nsamp) {
    lmax=nsamp-idetrend*mdetrend;
    isamp1=nsamp;
  }
  for (l=0;l<0.025*lmax;l++) {
    for (j=l;j<lmax-l;j++) {
      isamp=isamp0+j;
      if (isamp>=nsamp)
        break;
      if (j==l || ytmp[isamp+idm*nsamp]<ymin) {
        ymin=ytmp[isamp+idm*nsamp];
        isampmin=isamp;
      }
      if (j==l || ytmp[isamp+idm*nsamp]>ymax) {
        ymax=ytmp[isamp+idm*nsamp];
        isampmax=isamp;
      }
    }

    yswap=ytmp[isamp0+l+idm*nsamp];
    ytmp[isamp0+l+idm*nsamp]=ytmp[isampmin+idm*nsamp];
    ytmp[isampmin+idm*nsamp]=yswap;
    yswap=ytmp[isamp1-l-1+idm*nsamp];
    ytmp[isamp1-l-1+idm*nsamp]=ytmp[isampmax+idm*nsamp];
    ytmp[isampmax+idm*nsamp]=yswap;

    // Adjust sum
    syy-=ymin*ymin+ymax*ymax;
    s-=2.0;
  }
  ystd=1.148*sqrt(syy/s);

  // Normalize
  for (j=0;j<mdetrend;j++) {
    isamp=idetrend*mdetrend+j;
    if (isamp>=nsamp)
      break;
    y[isamp+idm*nsamp]/=ystd;
  }

  return;
}

void usage(void)
{

  printf("usage:  single_pulse_search.py [options] .dat files _or_ .singlepulse files\n");
  printf(" [-h  ]    : Display this help\n");
  printf(" [-m  ]    : Set the max downsampling in sec (see below for default)\n");
  printf(" [-t  ]    : Set a different threshold SNR (default=5.0)\n");

  return;
}

// Read a line of maximum length int lim from file FILE into string s
int fgetline(FILE *file,char *s,int lim)
{
  int c,i=0;

  while (--lim > 0 && (c=fgetc(file)) != EOF && c != '\n')
    s[i++] = c;
  if (c == '\n')
    s[i++] = c;
  s[i] = '\0';
  return i;
}

int read_info_file(char *fname,float *dt,float *dm,int *n)
{
  FILE *file;
  char line[LIM];
  int flag=0;

  // Open file
  file=fopen(fname,"r");
  if (file==NULL) {
    fprintf(stderr,"Error opening %s\n",fname);
    return -1;
  }

  // Loop over file contents
  while (fgetline(file,line,LIM)>0) {
    // Find sample time
    if (strstr(line,"Width of each time series bin (sec)")!=NULL) {
      sscanf(line+43,"%f",dt);
      flag++;
    }

    // Dispersion measure
    if (strstr(line,"Dispersion measure (cm-3 pc)")!=NULL) {
      sscanf(line+43,"%f",dm);
      flag++;
    }

    // Number of samples
    if (strstr(line,"Number of bins in the time series")!=NULL) {
      sscanf(line+43,"%d",n);
      flag++;
    }
  }
  
  // Close file
  fclose(file);

  // Error if information not found
  if (flag!=3) {
    fprintf(stderr,"Dispersion measure, sampling time or number of samples keywords not found in %s\n",fname);
    return -1;
  }

  return 0;
}

int main(int argc,char *argv[])
{
  int i,j,k,nx,mx,my,ny,nsamp=0,m,mdetrend,ndetrend,ndm=2;
  hipfftHandle ftr2cx,ftr2cy,ftc2rz;
  float *x,*y,*z,*dxs,*dzs,*fbuf;
  hipfftReal *dx,*dy,*dz;
  hipfftComplex *dcx,*dcy,*dcz;
  int *dmask,*mask,*dw,*w;
  int idist,odist,iembed,oembed,istride,ostride;
  int ds[]={2,3,4,6,9,14,20,30,45,70,100,150},dsmax=30;
  FILE *file;
  float dt=0.0001,dm=0.0,sigma=5.0,wmax=0.0;
  char *datfname,*inffname,*spfname;
  int arg=0,len;
  dim3 gridsize,blocksize;

  // Decode options
  if (argc>1) {
    while ((arg=getopt(argc,argv,"hm:t:"))!=-1) {
      switch(arg) {
	
      case 't':
	sigma=atof(optarg);
	break;
	
      case 'm':
	wmax=atof(optarg);
	break;

      case 'h':
        usage();
        return 0;

      default:
        usage();
	return 0;
      }
    }
  } else {
    usage();
    return 0;
  }

  // Set filenames
  len=strlen(argv[optind]);
  datfname=(char *) malloc(sizeof(char)*(len+2));
  inffname=(char *) malloc(sizeof(char)*(len+2));
  spfname=(char *) malloc(sizeof(char)*(len+15));

  // Assuming timeseries filename ends in .dat
  strcpy(datfname,argv[optind]);
  argv[optind][len-4]='\0';
  sprintf(inffname,"%s.inf",argv[optind]);
  sprintf(spfname,"%s.singlepulse",argv[optind]);

  // Read inf file
  if (read_info_file(inffname,&dt,&dm,&nsamp)!=0)
    return -1;

  // Allocate signal timeseries
  x=(float *) malloc(sizeof(float)*nsamp*ndm);
  z=(float *) malloc(sizeof(float)*nsamp*ndm);
  fbuf=(float *) malloc(sizeof(float)*nsamp);

  // Open file
  file=fopen(datfname,"r");
  if (file==NULL) {
    fprintf(stderr,"Error opening %s\n",datfname);
    return -1;
  }

  // Read buffer
  fread(fbuf,sizeof(float),nsamp,file);

  // Close file
  fclose(file);

  // Copy buffer
  for (i=0;i<ndm;i++)
    memcpy(x+i*nsamp,fbuf,sizeof(float)*nsamp);

  // Find number of kernels to convolve
  if (wmax>0.0) {
    for (i=1;i<sizeof(ds)/sizeof(ds[0]);i++) 
      if (ds[i-1]*dt<wmax && ds[i]*dt>wmax)
	break;
    my=i;
  } else {
    for (i=1;i<sizeof(ds)/sizeof(ds[0]);i++) 
      if (ds[i-1]<=dsmax && ds[i]>dsmax)
	break;
    my=i;
  }
   
  // Sizes
  m=8000;
  mdetrend=1000;
  nx=8192;
  mx=nx/2+1;

  // Number of FFTs
  ny=(int) ceil(nsamp/(float) m);

  // Number of detrend lengths
  ndetrend=(int) ceil(nsamp/(float) mdetrend);

  printf("%d samples, %d point fft, %d ffts, %d kernels, %d detrend blocks of %d samples\n",nsamp,nx,ny,my,ndetrend,mdetrend);

  // Allocate device memory for signal
  checkCudaErrors(hipMalloc((void **) &dxs,sizeof(float)*nsamp*ndm));
  checkCudaErrors(hipMalloc((void **) &dzs,sizeof(float)*nsamp*ndm));
  checkCudaErrors(hipMemcpy(dxs,x,sizeof(float)*nsamp*ndm,hipMemcpyHostToDevice));

  // Detrend timeseries
  blocksize.x=32;blocksize.y=32;blocksize.z=1;
  gridsize.x=ndetrend/blocksize.x+1;gridsize.y=ndm/blocksize.y+1;gridsize.z=1;
  detrend_and_normalize<<<gridsize,blocksize>>>(dxs,dzs,nsamp,mdetrend,ndetrend,ndm);

  // Allocate memory for padded signal
  checkCudaErrors(hipMalloc((void **) &dx,sizeof(hipfftReal)*nx*ny));

  // Padd signal
  blocksize.x=32;blocksize.y=32;blocksize.z=1;
  gridsize.x=nx/blocksize.x+1;gridsize.y=ny/blocksize.y+1;gridsize.z=1;
  padd_data<<<gridsize,blocksize>>>(dxs,dx,nsamp,nx,m,ny);

  // Allocate device memory
  y=(hipfftReal *) malloc(sizeof(hipfftReal)*nx*my);
  checkCudaErrors(hipMalloc((void **) &dy,sizeof(hipfftReal)*nx*my));
  checkCudaErrors(hipMalloc((void **) &dz,sizeof(hipfftReal)*nx*ny));
  checkCudaErrors(hipMalloc((void **) &dcx,sizeof(hipfftComplex)*mx*ny));
  checkCudaErrors(hipMalloc((void **) &dcy,sizeof(hipfftComplex)*mx*my));
  checkCudaErrors(hipMalloc((void **) &dcz,sizeof(hipfftComplex)*mx*ny));

  // Fill kernel
  for (j=0;j<my;j++) {
    for (i=0;i<nx;i++)
      y[j*nx+i]=0.0;

    if (ds[j]%2!=0) {
      // Odd factors
      for (i=0;i<ds[j]/2+1;i++)
        y[j*nx+i]+=1.0;
      for (i=nx-ds[j]/2;i<nx;i++)
        y[j*nx+i]+=1.0;
    } else {
      // Even factors
      for (i=0;i<ds[j]/2+1;i++)
        y[j*nx+i]+=1.0;
      if (ds[j]>2)
        for (i=nx-ds[j]/2+1;i<nx;i++)
          y[j*nx+i]+=1.0;
    }
    // Divide by sqrt(width)
    for (i=0;i<nx;i++)
      y[j*nx+i]/=sqrt(ds[j]);
  }

  // Copy kernels to device
  checkCudaErrors(hipMemcpy(dy,y,sizeof(hipfftReal)*nx*my,hipMemcpyHostToDevice));

  // Plan and FFT signal
  idist=nx;  odist=mx;  iembed=nx;  oembed=nx;  istride=1;  ostride=1;
  checkCudaErrors(hipfftPlanMany(&ftr2cx,1,&nx,&iembed,istride,idist,&oembed,ostride,odist,HIPFFT_R2C,ny));
  checkCudaErrors(hipfftExecR2C(ftr2cx,(hipfftReal *) dx,(hipfftComplex *) dcx));

  // Plan and FFT window
  idist=nx;  odist=mx;  iembed=nx;  oembed=nx;  istride=1;  ostride=1;
  checkCudaErrors(hipfftPlanMany(&ftr2cy,1,&nx,&iembed,istride,idist,&oembed,ostride,odist,HIPFFT_R2C,my));
  checkCudaErrors(hipfftExecR2C(ftr2cy,(hipfftReal *) dy,(hipfftComplex *) dcy));

  // Free input arrays
  hipFree(dx);
  hipFree(dy);

  // Plan convolved signal
  idist=mx;  odist=nx;  iembed=mx;  oembed=mx;  istride=1;  ostride=1;
  checkCudaErrors(hipfftPlanMany(&ftc2rz,1,&nx,&iembed,istride,idist,&oembed,ostride,odist,HIPFFT_C2R,ny));

  // Allocate mask
  mask=(int *) malloc(sizeof(int)*nsamp);
  checkCudaErrors(hipMalloc((void **) &dmask,sizeof(int)*nsamp));

  // Allocate width
  w=(int *) malloc(sizeof(int)*nsamp);
  checkCudaErrors(hipMalloc((void **) &dw,sizeof(int)*nsamp));

  // Set width
  for (i=0;i<nsamp;i++)
    w[i]=1;

  // Copy to device
  checkCudaErrors(hipMemcpy(dw,w,sizeof(int)*nsamp,hipMemcpyHostToDevice));

  // Loop over kernels
  for (k=0;k<my;k++) {
    // Complex multiplication
    PointwiseComplexMultiply<<<256,256>>>(dcx,dcy,dcz,mx*ny,mx,k,1.0/(float) nx);
 
    // FFT convolved signal
    checkCudaErrors(hipfftExecC2R(ftc2rz,(hipfftComplex *) dcz,(hipfftReal *) dz));

    // Unpadd convolved signal
    unpadd_data<<<256,256>>>(dzs,dz,nsamp,nx,m,ny);

    // Prune results
    prune<<<256,256>>>(dzs,nsamp,ds[k],dmask,sigma);

    // Store
    store<<<256,256>>>(dxs,dzs,dmask,dw,ds[k],nsamp);
  }

  // Prune final results
  prune_final<<<256,256>>>(dxs,dw,dmask,nsamp,sigma);

  // Store final results
  store_final<<<256,256>>>(dxs,dmask,nsamp,sigma);

  // Copy convolved signal to host
  checkCudaErrors(hipMemcpy(z,dxs,sizeof(hipfftReal)*nsamp,hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(w,dw,sizeof(int)*nsamp,hipMemcpyDeviceToHost));

  // Open single pulse file
  file=fopen(spfname,"w");
  if (file==NULL) {
    fprintf(stderr,"Error opening %s\n",spfname);
    return -1;
  }
  // Print results
  fprintf(file,"# DM      Sigma      Time (s)     Sample    Downfact   Sampling (s)\n");
  for (i=0,j=0;i<nsamp;i++) {
    if (z[i]>sigma) {
      fprintf(file,"%7.2f %7.2f %13.6f %10d     %3d   %g\n",dm,z[i],i*dt,i,w[i],dt);
      j++;
    }
  }
  fclose(file);
  printf("Found %d candidates\n",j);

  // Destroy plans
  hipfftDestroy(ftr2cx);
  hipfftDestroy(ftr2cy);
  hipfftDestroy(ftc2rz);

  // Free memory
  free(x);
  free(y);
  free(z);
  free(w);
  free(mask);
  hipFree(dz);
  hipFree(dcx);
  hipFree(dcy);
  hipFree(dcz);
  hipFree(dxs);
  hipFree(dw);
  hipFree(dmask);
  free(datfname);
  free(inffname);
  free(spfname);

  return 0;
}
